#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"

#define BLOCKS_PER_SM 1
#define SHARED_PCT 0.0
#define SM 142

__global__ void fillShared(int *blockCounter, int *flag) {
	// ensure all blocks have loaded their portion of shared memory
	atomicAdd(blockCounter, 1);

	// ensure updated blockCounter is visible across blocks
	__threadfence();

	// tell CPU that shared memory is fully saturated
	if (blockIdx.x == 0) {
		while (*blockCounter < gridDim.x) {
			// wait for all blocks to load shared memory
			__threadfence();
		}
		__threadfence();
		*flag = 0;
	}

	// spin with desired shared memory usage
	while (1) { }
}

int main() {
	// Ensure the opt in value is correct
	hipDeviceProp_t prop;
	int device = 0;
	SAFE(hipGetDeviceProperties(&prop, device));
	printf("Device: %s\nMax shared memory size opt in: %lu\n", prop.name, prop.sharedMemPerBlockOptin);

	// flag for CPU synchronization
	int *flag;
	SAFE(hipHostAlloc(&flag, sizeof(int), hipHostMallocMapped));
	*flag = 1;

	// block counter for signaing when shared memory caches are saturated
	int *blockCounter;
	SAFE(hipMalloc(&blockCounter, sizeof(int)));
	SAFE(hipMemset(blockCounter, 0, sizeof(int)));

	// Adjust shared partition to match usage
	SAFE(hipFuncSetAttribute(reinterpret_cast<const void*>(fillShared), hipFuncAttributePreferredSharedMemoryCarveout, SHARED_PCT));

	// Confirm shared carevout
	hipFuncAttributes attr;
	SAFE(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(fillShared)));
	printf("Carveout set to %d%% \n", attr.preferredShmemCarveout);

	// launch kernel and spin
	fillShared<<<SM * BLOCKS_PER_SM, 1>>>(blockCounter, flag);
	printf("Kernel launched - waiting for cache saturation...\n");
	while (*flag) { /* wait for caches to saturate */ }
	printf("Spinning!\n");
	SAFE(hipDeviceSynchronize());

	return 0;
}



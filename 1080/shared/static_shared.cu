#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"

#define BLOCKS_PER_SM 1
#define NUMBER 8
#define SHARED_PCT 51.51
#define SHARED_MEM_TB (int) (101376.0 * (SHARED_PCT / 100.0))
#define ELEMENTS SHARED_MEM_TB / 4
#define SM 142
#define STATIC_ELEMENTS 12288

__global__ void fillShared(int *blockCounter, int *flag) {
	__shared__ int static_array[STATIC_ELEMENTS];
	extern __shared__ float shared_array[];
	for (int i=0; i<ELEMENTS; i++) {
		shared_array[i] = NUMBER;
	}
	for (int i=0; i<STATIC_ELEMENTS; i++) {
		static_array[i] = NUMBER;
	}
	shared_array[0] += static_array[8];

	// ensure all blocks have loaded their portion of shared memory
	atomicAdd(blockCounter, 1);

	// ensure updated blockCounter is visible across blocks
	__threadfence();

	// tell CPU that shared memory is fully saturated
	if (blockIdx.x == 0) {
		while (*blockCounter < gridDim.x) {
			// wait for all blocks to load shared memory
			__threadfence();
		}
		__threadfence();
		*flag = 0;
	}

	// spin with desired shared memory usage
	while (0) { }
}

int main() {
	// Check the opt in value for dynamic shared memory
	hipDeviceProp_t prop;
	int device = 0;
	SAFE(hipGetDeviceProperties(&prop, device));
	printf("Device: %s\nMax shared memory size opt in: %lu\n", prop.name, prop.sharedMemPerBlockOptin);

	// print shared array size per TB/SM
	printf("Every TB is addressing a shared memory array of size %d\n", ELEMENTS);

	// flag for CPU synchronization
	int *flag;
	SAFE(hipHostAlloc(&flag, sizeof(int), hipHostMallocMapped));
	*flag = 1;

	// block counter for signaing when shared memory caches are saturated
	int *blockCounter;
	SAFE(hipMalloc(&blockCounter, sizeof(int)));
	SAFE(hipMemset(blockCounter, 0, sizeof(int)));

	// Allow TB to address dynamic shared memory as needed
	printf("Setting TB shared memory max to %d bytes\n", SHARED_MEM_TB);
	SAFE(hipFuncSetAttribute(reinterpret_cast<const void*>(fillShared), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM_TB));

	// Adjust shared partition to match usage
	SAFE(hipFuncSetAttribute(reinterpret_cast<const void*>(fillShared), hipFuncAttributePreferredSharedMemoryCarveout, SHARED_PCT));

	// Confirm shared carevout
	hipFuncAttributes attr;
	SAFE(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(fillShared)));
	printf("Carveout set to %d%% \n", attr.preferredShmemCarveout);
	printf("Static shared memory size: %lu\n", attr.sharedSizeBytes);

	// launch kernel and spin
	fillShared<<<SM * BLOCKS_PER_SM, 1, SHARED_MEM_TB>>>(blockCounter, flag);
	printf("Kernel launched - waiting for cache saturation...\n");
	while (*flag) { /* wait for caches to saturate */ }
	printf("Shared memory caches are saturated!\n");
	SAFE(hipDeviceSynchronize());

	return 0;
}



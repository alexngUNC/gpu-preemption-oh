#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess) {
        fprintf(stderr, "Error getting device count: %s\n", hipGetErrorString(error));
        return -1;
    }

    printf("Number of CUDA devices: %d\n", deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        error = hipGetDeviceProperties(&deviceProp, i);

        if (error != hipSuccess) {
            fprintf(stderr, "Error getting device properties for device %d: %s\n", i, hipGetErrorString(error));
            return -1;
        }

        printf("Device %d: %s\n", i, deviceProp.name);
	printf("Total Global memory: %zu\n", deviceProp.totalGlobalMem);
    }

    return 0;
}


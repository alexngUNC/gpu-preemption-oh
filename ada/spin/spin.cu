#include "hip/hip_runtime.h"
#include "testbench.h"
#include <stdio.h>
__global__ void spin() {
	while (1) {}
}

int main() {
	spin<<<1, 1>>>();
	SAFE(hipDeviceSynchronize());
	return 0;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"
#define CONSTANT_PCT 100.0
#define CONSTANT_MEMORY (int) (65536.0 * (CONSTANT_PCT / 100.0))
#define LENGTH CONSTANT_MEMORY / 4

__constant__ int constantData[LENGTH];

/*
1. Saturate every constant cache (launch max resident threads for whole GPU)
   May want to distribute cache access such that each thread of each thread block is accessing part; only the threqad block in aggregate accesses the whole buffer
2. Continually access constant cache in a loop
   e.g. loop update condition is i = (i+1)%LENGTH, and remove continuation condition
3. May want to do something to mitigate/reduce global loads
   e.g make the buffer no larger than the constant cache size
     Could keep shrinking the buffer until profiler shows few/no global loads in the steady-state:q

*/

__global__ void readConstant(int *result, int *flag) {
	int sum = 0;
	for (int i=0; i<LENGTH; i++) {
		sum += constantData[i];
	}
	*result = sum;
	*flag = 0;
	while (0) { }
}

int
main()
{
	// print cache percent info
	printf("Constant cache usage: %.1f | %d bytes | %d elements\n", CONSTANT_PCT, CONSTANT_MEMORY, LENGTH);

	// allocate host data for constant cache
	int hostData[LENGTH];
	for (int i=0; i<LENGTH; i++) {
		hostData[i] = 2;
	}

	// copy data to constant memory cache
	SAFE(hipMemcpyToSymbol(HIP_SYMBOL(constantData), hostData, CONSTANT_MEMORY));

	// flag for synchronization
	int *flag;
	SAFE(hipHostAlloc(&flag, sizeof(int), hipHostMallocMapped));
	*flag = 1;

	// result ptr for sum
	int *result;
	SAFE(hipHostAlloc(&result, sizeof(int), hipHostMallocMapped));
	*result = 0;

	// spin on GPU once constant memory is accessed
	readConstant<<<1, 1>>>(result, flag);
	while (*flag) {}
	printf("Constant memory has been read!\n");
	SAFE(hipDeviceSynchronize());

	// print result
	printf("Total: %d\n", *result);

	return 0;
}

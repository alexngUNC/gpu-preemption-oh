#include "hip/hip_runtime.h"
/* Copyright 2021-2023 Joshua Bakita
 * Simple kernel that spins on the GPU for a specified number of iterations,
 * while tracking and printing the necessary CPU time.
 */
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "testbench.h"

__global__ void loop_on_gpu(unsigned long iters, int *__unused) {
	for (volatile int i = 0; i < iters; i++) (*__unused)++;
}

int main(int argc, char **argv) {
	int res, *__unused;
	struct timespec start, end;

	if (argc != 2 || !strcmp(argv[1], "--help") || !strcmp(argv[1], "-h")) {
		fprintf(stderr, "Usage: %s <# of millions of iterations, or -1 for infinite>\n",
		        argv[0]);
		return 1;
	}

	// Input is multiplied by one million, unless infinite
	unsigned long num_iters = strtoul(argv[1], NULL, 10);
	if (num_iters != (unsigned long)(-1))
		num_iters *= 1000 * 1000;

	// Initialize CUDA and a context (hack)
	SAFE(hipMalloc(&__unused, 8));

	// Run iterations on a single thread
	clock_gettime(CLOCK_MONOTONIC_RAW, &start);
	loop_on_gpu<<<1,1>>>(num_iters, __unused);
	SAFE(hipGetLastError() /* Check successful launch */);
	SAFE(hipDeviceSynchronize());
	clock_gettime(CLOCK_MONOTONIC_RAW, &end);

	// Print detailed timing information
	long elapsed = timediff(start, end);
	fprintf(stderr, "Started at %ld ns, ended at %ld ns\n",
	        s2ns(start.tv_sec) + start.tv_nsec, s2ns(end.tv_sec) + end.tv_nsec);
	fprintf(stderr, "%ld ns (%.2f ms) elapsed\n", elapsed, elapsed / (1000 * 1000.));

	// Verify success (fool optimizer)
	SAFE(hipMemcpy(&res, __unused, 8, hipMemcpyDeviceToHost));
	// Theoretically this can happen if `__unused` wraps around; maybe for a
	// very small `long` type, or a very long run. More likely indicates an error.
	if (!res)
		fprintf(stderr, "CRITICAL: Zero iterations seem completed. Likely incorrect "
		        "arguments, internal error, or corruption of CUDA internal state.\n");

	return 0;
}

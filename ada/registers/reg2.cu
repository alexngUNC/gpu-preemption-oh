#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"

#define BLOCKS_PER_SM 6
#define LENGTH 100
#define TB_SIZE 256

__global__ void kernel(int *data, bool print) {
	int sdata[LENGTH];
	int *pdata[LENGTH];
	// put int ptrs in array
	for (int i=0; i<LENGTH/4; i++) {
		pdata[i] = data;
		sdata[i] = *pdata[i];
	}

	// pull data from int ptrs
	for (int i=0; i<LENGTH; i++) {
		sdata[i] = *pdata[i];
	}

	// fake print
	for (int i=0; i<LENGTH; i++) {
    		if (print) printf("sdata[%d] = %d\n", i, sdata[i]);
	}
}

int main() {
	// check which GPU is being used
	hipDeviceProp_t deviceProp;
	SAFE(hipGetDeviceProperties(&deviceProp, 0));
	printf("----- Device: %s -----\n", deviceProp.name);
	int sm_count = deviceProp.multiProcessorCount;
	printf(" SM Count: %d\n", sm_count);
  	int *data;
	//SAFE(hipHostAlloc(&data, sizeof(float), hipHostMallocMapped));
  	SAFE(hipMalloc(&data, sizeof(int)));
  	SAFE(hipMemset(data, 0, sizeof(int)));
	// Create CUDA events for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Record the start event
	hipEventRecord(start);

  	//kernel<<<sm_count * BLOCKS_PER_SM, TB_SIZE>>>(data, false);
  	kernel<<<sm_count * BLOCKS_PER_SM, TB_SIZE>>>(data, false);
  	SAFE(hipDeviceSynchronize());

	// Record the stop event
	hipEventRecord(stop);
  	SAFE(hipDeviceSynchronize());

	// Calculate the elapsed time
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	// Output the elapsed time
	printf("----- Elapsed time: %f ms -----\n", milliseconds);
  	return 0;
}

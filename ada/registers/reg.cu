#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"

#define BLOCKS_PER_SM 6
#define LENGTH 1
#define TB_SIZE 256

template <typename T>
__global__ void kernel(T *data, bool loop) {
	T sdata[LENGTH];
	while (1) {
		for (int i = 0; i < LENGTH; i++) {
			sdata[i] = data[i];
		}
		for (int i=0; i<LENGTH; i++) {
    			if (loop) printf("sdata[%d] = %d\n", i, sdata[i]);
		}
	}
}

int main() {
	// check which GPU is being used
	hipDeviceProp_t deviceProp;
	SAFE(hipGetDeviceProperties(&deviceProp, 0));
	printf("----- Device: %s -----\n", deviceProp.name);
	int sm_count = deviceProp.multiProcessorCount;
	printf(" SM Count: %d\n", sm_count);
  	int *data;
	//SAFE(hipHostAlloc(&data, sizeof(float), hipHostMallocMapped));
  	SAFE(hipMalloc(&data, LENGTH*sizeof(float)));
  	SAFE(hipMemset(data, 0, LENGTH*sizeof(float)));
	// Create CUDA events for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Record the start event
	hipEventRecord(start);

  	//kernel<<<sm_count * BLOCKS_PER_SM, TB_SIZE>>>(data, false);
  	kernel<<<sm_count * BLOCKS_PER_SM, TB_SIZE>>>(data, false);
  	SAFE(hipDeviceSynchronize());

	// Record the stop event
	hipEventRecord(stop);
  	SAFE(hipDeviceSynchronize());

	// Calculate the elapsed time
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	// Output the elapsed time
	printf("----- Elapsed time: %f ms -----\n", milliseconds);
  	return 0;
}

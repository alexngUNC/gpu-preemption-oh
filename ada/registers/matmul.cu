#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"

#define BLOCKS_PER_SM 3 
#define LENGTH 50
#define TB_SIZE 512

__global__ void kernel(float *data, bool loop) {
	int a[LENGTH * LENGTH];
	int b[LENGTH * LENGTH];
	int c[LENGTH * LENGTH];
	int temp = 0;
	for (int i=0; i<LENGTH; i++) {
		for (int j=0; j<LENGTH; j++) {
			c[i*LENGTH + j] += a[i*LENGTH + j] * b[j*LENGTH + i];	
			temp += c[i*LENGTH + j];
		}
	}
	*data = temp;
}

int main() {
	// check which GPU is being used
	hipDeviceProp_t deviceProp;
	SAFE(hipGetDeviceProperties(&deviceProp, 0));
	printf("----- Device: %s -----\n", deviceProp.name);
	int sm_count = deviceProp.multiProcessorCount;
	printf(" SM Count: %d\n", sm_count);
  	float *data;
	//SAFE(hipHostAlloc(&data, sizeof(float), hipHostMallocMapped));
  	SAFE(hipMalloc(&data, LENGTH*sizeof(float)));
  	SAFE(hipMemset(data, 0, LENGTH*sizeof(float)));
	// Create CUDA events for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Record the start event
	hipEventRecord(start);

  	//kernel<<<sm_count * BLOCKS_PER_SM, TB_SIZE>>>(data, false);
  	kernel<<<sm_count * BLOCKS_PER_SM, TB_SIZE>>>(data, false);
  	SAFE(hipDeviceSynchronize());

	// Record the stop event
	hipEventRecord(stop);
  	SAFE(hipDeviceSynchronize());

	// Calculate the elapsed time
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	// Output the elapsed time
	printf("----- Elapsed time: %f ms -----\n", milliseconds);
  	return 0;
}
